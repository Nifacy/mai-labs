#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

typedef unsigned char uchar;

#define sqr3(x) ((x)*(x)*(x))
#define sqr(x) ((x)*(x))


struct t_item {
	float x;
	float y;
	float z;
	float dx;
	float dy;
	float dz;
	float q;
};

t_item item;


int w = 1024, h = 648;

float x = -1.5, y = -1.5, z = 1.0;
float dx = 0.0, dy = 0.0, dz = 0.0;
float yaw = 0.0, pitch = 0.0;
float dyaw = 0.0, dpitch = 0.0;

float speed = 0.2;

const float a2 = 15.0;	
const int np = 100;

GLUquadric* quadratic;	

hipGraphicsResource *res;
GLuint textures[2];	
GLuint vbo;	



void display() {
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	
	gluPerspective(90.0f, (GLfloat)w/(GLfloat)h, 0.1f, 100.0f);
	
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	gluLookAt(x, y, z,
				x + cos(yaw) * cos(pitch),
				y + sin(yaw) * cos(pitch),
				z + sin(pitch),
				0.0f, 0.0f, 1.0f);
	
	glBindTexture(GL_TEXTURE_2D, textures[0]);
	static float angle = 0.0;
	
	glPushMatrix();
		glTranslatef(item.x, item.y, item.z);
		glRotatef(angle, 0.0, 0.0, 1.0);
		gluSphere(quadratic, 2.5f, 32, 32);
	glPopMatrix();
	angle += 5.0;
	
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, vbo);
	glBindTexture(GL_TEXTURE_2D, textures[1]);	
	glTexImage2D(GL_TEXTURE_2D, 0, 3, (GLsizei)np, (GLsizei)np, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL); 
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
	
	glBegin(GL_QUADS);			// Рисуем пол
		glTexCoord2f(0.0, 0.0);
		glVertex3f(-a2, -a2, 0.0);

		glTexCoord2f(1.0, 0.0);
		glVertex3f(a2, -a2, 0.0);

		glTexCoord2f(1.0, 1.0);
		glVertex3f(a2, a2, 0.0);

		glTexCoord2f(0.0, 1.0);
		glVertex3f(-a2, a2, 0.0);
	glEnd();
	
	glBindTexture(GL_TEXTURE_2D, 0);
	
	glLineWidth(2);								// Толщина линий				
	glColor3f(0.5f, 0.5f, 0.5f);				// Цвет линий
	glBegin(GL_LINES);							// Последующие пары вершин будут задавать линии
		glVertex3f(-a2, -a2, 0.0);
		glVertex3f(-a2, -a2, 2.0 * a2);

		glVertex3f(a2, -a2, 0.0);
		glVertex3f(a2, -a2, 2.0 * a2);

		glVertex3f(a2, a2, 0.0);
		glVertex3f(a2, a2, 2.0 * a2);

		glVertex3f(-a2, a2, 0.0);
		glVertex3f(-a2, a2, 2.0 * a2);
	glEnd();

	glBegin(GL_LINE_LOOP);						// Все последующие точки будут соеденены замкнутой линией
		glVertex3f(-a2, -a2, 0.0);
		glVertex3f(a2, -a2, 0.0);
		glVertex3f(a2, a2, 0.0);
		glVertex3f(-a2, a2, 0.0);
	glEnd();

	glBegin(GL_LINE_LOOP);
		glVertex3f(-a2, -a2, 2.0 * a2);
		glVertex3f(a2, -a2, 2.0 * a2);
		glVertex3f(a2, a2, 2.0 * a2);
		glVertex3f(-a2, a2, 2.0 * a2);
	glEnd();

	glColor3f(1.0f, 1.0f, 1.0f);

	glutSwapBuffers();
}


__global__ void kernel(uchar4 *data, t_item item, float t) {	// Генерация текстуры пола на GPU
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int i, j;
	float x, y, fg, fb;
	for(i = idx; i < np; i += offsetx)
		for(j = idy; j < np; j += offsety) {
			x = (2.0 * i / (np - 1.0) - 1.0) * a2;
			y = (2.0 * j / (np - 1.0) - 1.0) * a2;
			fb = 100.0 * (sin(0.1 * x * x + t) + cos(0.1 * y * y + t * 0.6) + sin(0.1 * x * x + 0.1 * y * y + t * 0.3));
			fg = 10000.0 * item.q / (sqr(x - item.x) + sqr(y - item.y) + sqr(item.z) + 0.001);
			fg = min(max(0.0f, fg), 255.0f);
			fb = min(max(0.0f, fb), 255.0f);
			data[j * np + i] = make_uchar4(0, (int)fg, (int)fb, 255);
		}
}


void update() {
//	dz -= 0.0001;

	float v = sqrt(dx * dx + dy * dy + dz * dz);
	if (v > speed) {		// Ограничение максимальной скорости
		dx *= speed / v;
		dy *= speed / v;
		dz *= speed / v;
	}
	x += dx; dx *= 0.99;
	y += dy; dy *= 0.99;
	z += dz; dz *= 0.99;

	if (z < 1.0) {
		z = 1.0;
		dz = 0.0;
	}
	
	if (fabs(dpitch) + fabs(dyaw) > 0.0001) {
		yaw += dyaw;
		pitch += dpitch;
		pitch = min(M_PI / 2.0 - 0.0001, max(-M_PI / 2.0 + 0.0001, pitch));
		dyaw *= 0.5;
		dpitch *= 0.5;
		//dyaw = dpitch = 0.0;
	}
	
	float w = 0.999999, e0 = 1e-3, dt = 0.1, K = 50.0;
	
	item.dx *= w;
	item.dy *= w;
	item.dz *= w;
	
	item.dx += item.q * item.q * K * (item.x - a2) / (sqr3(fabs(item.x - a2)) + e0) * dt;
	item.dx += item.q * item.q * K * (item.x + a2) / (sqr3(fabs(item.x + a2)) + e0) * dt;

	item.dy += item.q * item.q * K * (item.y - a2) / (sqr3(fabs(item.y - a2)) + e0) * dt;
	item.dy += item.q * item.q * K * (item.y + a2) / (sqr3(fabs(item.y + a2)) + e0) * dt;

	item.dz += item.q * item.q * K * (item.z - 2.0 * a2) / (sqr3(fabs(item.z - 2.0 * a2)) + e0) * dt;
	item.dz += item.q * item.q * K * (item.z + 0.0) / (sqr3(fabs(item.z + 0.0)) + e0) * dt;
	
	float l = sqrt(sqr(item.x - x) + sqr(item.y - y) + sqr(item.z - z));
	
	item.dx += 3.0 * item.q * K * (item.x - x) / (l * l * l + e0) * dt;
	item.dy += 3.0 * item.q * K * (item.y - y) / (l * l * l + e0) * dt;
	item.dz += 3.0 * item.q * K * (item.z - z) / (l * l * l + e0) * dt;
	
	item.x += item.dx * dt;
	item.y += item.dy * dt;
	item.z += item.dz * dt;
	
	
	static float t = 0.0;
	uchar4* dev_data;
	size_t size;
	hipGraphicsMapResources(1, &res, 0);		// Делаем буфер доступным для CUDA
	hipGraphicsResourceGetMappedPointer((void**) &dev_data, &size, res);	// Получаем указатель на память буфера
	kernel<<<dim3(32, 32), dim3(32, 8)>>>(dev_data, item, t);		
	hipGraphicsUnmapResources(1, &res, 0);		// Возращаем буфер OpenGL'ю что бы он мог его использовать
	t += 0.01;
	
	glutPostRedisplay();
}


void keys(unsigned char key, int x, int y) {
	switch (key) {
		case 'w':                 // "W" Движение вперед
			dx += cos(yaw) * cos(pitch) * speed;
			dy += sin(yaw) * cos(pitch) * speed;
			dz += sin(pitch) * speed;
		break;
		case 's':                 // "S" Назад
			dx += -cos(yaw) * cos(pitch) * speed;
			dy += -sin(yaw) * cos(pitch) * speed;
			dz += -sin(pitch) * speed;
		break;
		case 'a':                 // "A" Влево
			dx += -sin(yaw) * speed;
			dy += cos(yaw) * speed;
			break;
		case 'd':                 // "D" Вправо
			dx += sin(yaw) * speed;
			dy += -cos(yaw) * speed;
		break;
		case 27:
			hipGraphicsUnregisterResource(res);
			glDeleteTextures(2, textures);
			glDeleteBuffers(1, &vbo);
			gluDeleteQuadric(quadratic);
			exit(0);
		break;
	}
}

void mouse(int x, int y) {
	static int x_prev = w / 2, y_prev = h / 2;
	float dx = 0.005 * (x - x_prev);
    float dy = 0.005 * (y - y_prev);
	dyaw -= dx;
    dpitch -= dy;
	x_prev = x;
	y_prev = y;
	if ((x < 20) || (y < 20) || (x > w - 20) || (y > h - 20)) {
		glutWarpPointer(w / 2, h / 2);
		x_prev = w / 2;
		y_prev = h / 2;
    }
}


void reshape(int w_new, int h_new) { 
	w = w_new;
	h = h_new;
	glViewport(0, 0, w, h); 
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
}


int main(int argc, char **argv) {
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowSize(w, h);
	glutCreateWindow("OpenGL");

	glutIdleFunc(update);
	glutDisplayFunc(display);
	glutKeyboardFunc(keys);
	glutPassiveMotionFunc(mouse);
	glutReshapeFunc(reshape);

	glutSetCursor(GLUT_CURSOR_NONE);
	
	int wt, ht;
	FILE *in = fopen("in.data", "rb");
	fread(&wt, sizeof(int), 1, in);
	fread(&ht, sizeof(int), 1, in);
	uchar *data = (uchar *)malloc(sizeof(uchar) * wt * ht * 4);
	fread(data, sizeof(uchar), 4 * wt * ht, in);
	fclose(in);

	glGenTextures(2, textures);
	glBindTexture(GL_TEXTURE_2D, textures[0]);
	glTexImage2D(GL_TEXTURE_2D, 0, 3, (GLsizei)wt, (GLsizei)ht, 0, GL_RGBA, GL_UNSIGNED_BYTE, (void*)data);
	free(data);

	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	
	quadratic = gluNewQuadric();
	gluQuadricTexture(quadratic, GL_TRUE);

	glBindTexture(GL_TEXTURE_2D, textures[1]);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	
	glEnable(GL_TEXTURE_2D);                             // Разрешить наложение текстуры
	glShadeModel(GL_SMOOTH);                             // Разрешение сглаженного закрашивания
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);                // Черный фон
	glClearDepth(1.0f);                                  // Установка буфера глубины
	glDepthFunc(GL_LEQUAL);                              // Тип теста глубины. 
	glEnable(GL_DEPTH_TEST);                			 // Включаем тест глубины
	glEnable(GL_CULL_FACE);                 			 // Режим при котором, тектуры накладываются только с одной стороны
	
	glewInit();	
	glGenBuffers(1, &vbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, vbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, np * np * sizeof(uchar4), NULL, GL_DYNAMIC_DRAW);
	hipGraphicsGLRegisterBuffer(&res, vbo, cudaGraphicsMapFlagsWriteDiscard);	
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
	
	item.x = item.y = item.z = 5.0;						
	item.dx = item.dy = item.dz = 0.1;
	item.q = 1.0;
	
	glutMainLoop();
}
