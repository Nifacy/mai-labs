#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>

#include "texture.cuh"
#include "texture_projection.cuh"
#include "utils.cuh"
#include "canvas.cuh"
#include "polygon.cuh"
#include "debug_render.cuh"
#include "cpu_render.cuh"
#include "render.cuh"
#include "scene.cuh"


__device__ Canvas::TColor VectorToColor2(Vector::TVector3 v) {
    return {
        (unsigned char) Max(0, Min(int(v.x * 255.0), 255)),
        (unsigned char) Max(0, Min(int(v.y * 255.0), 255)),
        (unsigned char) Max(0, Min(int(v.z * 255.0), 255)),
        255
    };
}

__device__ Vector::TVector3 ColorToVector2(Canvas::TColor color) {
    return Vector::Mult(
        1.0 / 255.0,
        { (double) color.r, (double) color.g, (double) color.b}
    );
}

__global__ void GpuSsaa(Canvas::TCanvas src, Canvas::TCanvas dst, unsigned int coef) {
    int startX = blockDim.x * blockIdx.x + threadIdx.x;
    int startY = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetX = blockDim.x * gridDim.x;
    int offsetY = blockDim.y * gridDim.y;

    for (unsigned int x = startX; x < dst.width; x += offsetX) {
        for (unsigned int y = startY; y < dst.height; y += offsetY) {
            Vector::TVector3 color = { 0.0, 0.0, 0.0 };

            for (unsigned int dx = 0; dx < coef; ++dx) {
                for (unsigned int dy = 0; dy < coef; ++dy) {
                    Canvas::TColor srcColor = Canvas::GetPixel(&src, { x * coef + dx, y * coef + dy });
                    color = Vector::Add(color, ColorToVector2(srcColor));
                }
            }

            color = Vector::Mult(1.0 / coef / coef, color);
            Canvas::PutPixel(&dst, { x, y }, VectorToColor2(color));
        }
    }
}

__global__ void kernel(
    TRay *current, int currentCount,
    TRay *next, int *cursor,
    Polygon::TPolygon *polygons, size_t polygonsAmount,
    TLight *lights, size_t lightsAmount,
    Canvas::TCanvas canvas,
    int *lock
) {
    size_t offset = gridDim.x * blockDim.x;
    size_t start = blockDim.x * blockIdx.x + threadIdx.x;

    for (int j = start; j < currentCount; j += offset) {
        TRay el = current[j];
        Canvas::TColor color = VectorToColor2(Ray(el, polygons, polygonsAmount, lights, lightsAmount, next, cursor, true));

        atomicAdd(lock, 1);
        Canvas::AddPixel(&canvas, { .x = el.pixelPos.x, .y = el.pixelPos.y }, color);
        atomicSub(lock, 1);
    }
}

__global__ void initRays(Canvas::TCanvas canvas, Vector::TVector3 pc, Vector::TVector3 pv, double angle, TRay *rays) {
    double dw = 2.0 / (canvas.width - 1.0);
    double dh = 2.0 / (canvas.height - 1.0);
    double z = 1.0 / tan(angle * M_PI / 360.0);

    Vector::TVector3 bz = Vector::Normalize(Vector::Sub(pv, pc));
    Vector::TVector3 bx = Vector::Normalize(Vector::Prod(bz, {0.0, 0.0, 1.0}));
    Vector::TVector3 by = Vector::Normalize(Vector::Prod(bx, bz));

    int startX = blockDim.x * blockIdx.x + threadIdx.x;
    int startY = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetX = blockDim.x * gridDim.x;
    int offsetY = blockDim.y * gridDim.y;

    for(unsigned int i = startX; i < canvas.width; i += offsetX) {
        for(unsigned int j = startY; j < canvas.height; j += offsetY) {
            Vector::TVector3 v = {-1.0 + dw * i, (-1.0 + dh * j) * canvas.height / canvas.width, z};
            Vector::TVector3 dir = Vector::Mult(bx, by, bz, v);
            rays[j * canvas.width + i] = {
                .pos = pc,
                .dir = Vector::Normalize(dir),
                .color = { 1.0, 1.0, 1.0 },
                .pixelPos = { .x = i, .y = canvas.height - 1 - j },
                .depth = 0
            };

            Canvas::PutPixel(&canvas, { .x = i, .y = canvas.height - 1 - j }, { 0, 0, 0, 255 });
		}
	}
}

void GpuRender2(Vector::TVector3 pc, Vector::TVector3 pv, double angle, Canvas::TCanvas *canvas, std::vector<Polygon::TPolygon> &polygons, std::vector<TLight> &lights) {
    size_t initialRayCount = canvas->width * canvas->height;

    TRay *rays1;
    SAVE_CUDA(hipMalloc((void**) &rays1, 8 * initialRayCount * sizeof(TRay)));
    initRays<<<100, 100>>>(*canvas, pc, pv, angle, rays1);
    hipDeviceSynchronize();
    SAVE_CUDA(hipGetLastError());

    int *rays1Count;
    SAVE_CUDA(hipMalloc((void **) &rays1Count, sizeof(int)));
    SAVE_CUDA(hipMemcpy(rays1Count, &initialRayCount, sizeof(int), hipMemcpyHostToDevice));

    int *lock;
    SAVE_CUDA(hipMalloc((void **) &lock, sizeof(int)));

    TRay *rays2;
    SAVE_CUDA(hipMalloc((void**) &rays2, 8 * initialRayCount * sizeof(TRay)));

    int *rays2Count;
    SAVE_CUDA(hipMalloc((void **) &rays2Count, sizeof(int)));
    SAVE_CUDA(hipMemset(rays2Count, 0, sizeof(int)));

    Polygon::TPolygon *devicePolygons;
    SAVE_CUDA(hipMalloc((void**) &devicePolygons, polygons.size() * sizeof(Polygon::TPolygon)));
    SAVE_CUDA(hipMemcpy(devicePolygons, polygons.data(), polygons.size() * sizeof(Polygon::TPolygon), hipMemcpyHostToDevice));

    TLight *deviceLights;
    SAVE_CUDA(hipMalloc((void**)& deviceLights, lights.size() * sizeof(TLight)));
    SAVE_CUDA(hipMemcpy(deviceLights, lights.data(), lights.size() * sizeof(TLight), hipMemcpyHostToDevice));

    int it = 0;

    for (int i = 0;; i = (i + 1) % 2) {
        TRay *current = (i % 2 == 0) ? rays1 : rays2;
        int *currentCount = (i % 2 == 0) ? rays1Count : rays2Count;
        TRay *next = (i % 2 == 0) ? rays2 : rays1;
        int *nextCount = (i % 2 == 0) ? rays2Count : rays1Count;

        int tmp;
        hipMemcpy(&tmp, currentCount, sizeof(int), hipMemcpyDeviceToHost);
        SAVE_CUDA(hipMemset(nextCount, 0, sizeof(int)));

        std::cout << "iteration: " << it << ", rays: " << tmp << std::endl;
        
        if (tmp == 0) {
            break;
        }

        kernel<<<200, 200>>>(
            current, tmp,
            next, nextCount,
            devicePolygons, polygons.size(),
            deviceLights, lights.size(),
            *canvas,
            lock
        );
        hipDeviceSynchronize();
        SAVE_CUDA(hipGetLastError());
        std::cout << "iteration: " << it << ", rays: " << tmp << " [ok]" << std::endl;

        it++;
    }

    hipFree(devicePolygons);
    hipFree(deviceLights);
    hipFree(rays1);
    hipFree(rays1Count);
    hipFree(rays2);
    hipFree(rays2Count);
    hipFree(lock);
}


__global__ void GpuDraw(Canvas::TCanvas canvas) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    for (unsigned int x = idx; x < canvas.width; x += offsetx) {
        for (unsigned int y = idy; y < canvas.height; y += offsety) {
            Canvas::PutPixel(&canvas, Canvas::TPosition { x, y }, Canvas::TColor { 255, 0, 0, 255 });
        }
    }
}

/* Cpu render */

void CpuDraw(Canvas::TCanvas canvas) {
    for (unsigned int x = 0; x < canvas.width; ++x) {
        for (unsigned int y = 0; y < canvas.height; ++y) {
            Canvas::PutPixel(&canvas, Canvas::TPosition { x, y }, Canvas::TColor { 255, 0, 0, 255 });
        }
    }
}


const unsigned int CANVAS_WIDTH = 400;
const unsigned int CANVAS_HEIGHT = 400;


int main(int argc, char *argv[]) {
    std::string deviceTypeArg = std::string(argv[1]);    
    Canvas::TCanvas canvas, extendedCanvas;

    DeviceType deviceType = (deviceTypeArg == "gpu") ? DeviceType::GPU : DeviceType::CPU;

    Canvas::Init(&canvas, CANVAS_WIDTH, CANVAS_HEIGHT, deviceType);
    Canvas::Init(&extendedCanvas, 2 * CANVAS_WIDTH, 2 * CANVAS_HEIGHT, deviceType);

    std::vector<Polygon::TPolygon> polygons;
    std::vector<TLight> lights = {
        { .position = { 5.0, 5.0, 5.0 }, .color = { 1.0, 1.0, 1.0 } }
    };

    build_space(polygons, deviceType);

    Vector::TVector3 pc = { 0.0, 6.0, 4.0 };
    Vector::TVector3 pv = { 0.0, -3.0, -1.2 };

    if (deviceTypeArg == "gpu") {
        std::cerr << "[log] using GPU render ..." << std::endl;

        GpuRender2(
            pc, pv, 120.0,
            &extendedCanvas,
            polygons, lights
        );

        GpuSsaa<<<200, 200>>>(extendedCanvas, canvas, 2);
        hipDeviceSynchronize();
        SAVE_CUDA(hipGetLastError());

    } else if (deviceTypeArg == "cpu") {
        std::cerr << "[log] using CPU render ..." << std::endl;

        render(
            pc, pv, 120.0,
            &extendedCanvas,
            polygons, lights
        );

        CpuSsaa(&extendedCanvas, &canvas, 2);

    } else {
        std::cerr << "[log] using debug render ..." << std::endl;
        DebugRenderer::Render(canvas, polygons.data(), polygons.size());
    }

    Canvas::Dump(&canvas, "build/0.data");
    Canvas::Destroy(&canvas);
    Canvas::Destroy(&extendedCanvas);

    return 0;
}
